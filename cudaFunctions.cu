#include "hip/hip_runtime.h"
#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "myFunctions.h"

__device__ int numOfChars(char* group){
    int count=0;
    while(*(group+count) != '\0'){
        count++;
    }
    return count;
}
__device__ int isEqual(char x, char y){
	if (x == y) {
			return 1;
		}
		return 0;
}

__device__ int isConservative(char x, char y) {
	char *conserGroups[9] = { "NDEQ", "NEQK", "STA", "MILV", "QHRK", "NHQK",
			"FYW", "HY", "MILF" };
	int count, groupSize = 0;
	for (int i = 0; i < 9; i++) {
		count = 0;
		groupSize = numOfChars(*(conserGroups + i));
		for (int j = 0; j < groupSize; j++) {

			if (isEqual(x, *(*(conserGroups + i) + j))) {
				count++;
			} else if (isEqual(y, *(*(conserGroups + i) + j))) {
				count++;
			}
		}
		if (count == 2) {
			return 1;
		}
	}
	return 0;
}

__device__ int isSemiConservative(char x, char y) {
	char *semiConserGroups[11] = { "SAG", "ATV", "CSA", "SGND", "STPA", "STNK",
			"NEQHRK", "NDEQHK", "SNDEQK", "HFY", "FVLIM" };
	int count, groupSize = 0;
	for (int i = 0; i < 11; i++) {
		count = 0;
		groupSize = numOfChars(*(semiConserGroups + i));
		for (int j = 0; j < groupSize; j++) {

			if (isEqual(x, *(*(semiConserGroups + i) + j))) {
				count++;
			} else if (isEqual(y, *(*(semiConserGroups + i) + j))) {
				count++;
			}
		}
		if (count == 2) {
			return 1;
		}
	}
	return 0;
}

__device__ double calcResult(double* results, double* weights) {

	double star = *(results);
	double colon = *(results + 1);
	double dot = *(results + 2);
	double space = *(results + 3);
	double score = star * weights[0] - colon * weights[1] - dot * weights[2] - space * weights[3];
	return score;
}
//each offset is calculating his score
__global__ void comparisonMutantPrime_gpu(char* primeSeq, char* mutant,
		int lenMutant, int offset, double* weights, double* offSetScores, int my_rank) {

	int tid = blockIdx.x * blockDim.x + threadIdx.x;//identify the current offset
	if(tid < lenMutant){
		char primeSeqChar;
		char mutantChar;
		double results[4] = {0, 0, 0, 0}; /*Sum each one of the equation variables {*, :, . ,""}*/
		
		for(int j=0;j<lenMutant;j++){
			primeSeqChar = *(primeSeq + tid + j);
			mutantChar = *(mutant + j);
			if (isEqual(primeSeqChar, mutantChar)) {
				//*
				results[0]++;
			} else if (isConservative(primeSeqChar, mutantChar)) {
				//:
				results[1]++;
			} else if (isSemiConservative(primeSeqChar, mutantChar)) {
				//.
				results[2]++;
			} else {
				//isSpace
				//" "
				results[3]++;
			}
		}
		double score = calcResult(results, weights);
		offSetScores[tid] = score;
	}
}

double ComputeOnGPU(char* primeSeq, char* mutant, double* weights, int possibleOffset, int tid, int my_rank, int* p_offset) {

	
	int lenPrimeSequence = strlen(primeSeq);
	int lenMutant = strlen(mutant);
	
	//Allocateing memory
	double *offsetScore = (double*)malloc(sizeof(double)*possibleOffset);
	double *d_offsetScore;
	hipMalloc((void**) &d_offsetScore, sizeof(double) * possibleOffset);
	
	//Allocateing memory and trasfer data from CPU to GPU
	char *d_primeSeq;
	hipMalloc((void**) &d_primeSeq, sizeof(char) * (lenPrimeSequence+1));
	hipMemcpy(d_primeSeq, primeSeq, sizeof(char) * (lenPrimeSequence+1),
			hipMemcpyHostToDevice);

	//Allocateing memory and trasfer data from CPU to GPU
	char* d_mutant;
	hipMalloc((void**) &d_mutant, sizeof(char) * (lenMutant+1));
	hipMemcpy(d_mutant, mutant, sizeof(char) * (lenMutant+1),
			hipMemcpyHostToDevice);

	//Allocateing memory and trasfer data from CPU to GPU
	double *d_weights;
	hipMalloc((void**) &d_weights, sizeof(double) * EQUATION_ELEMENTS);
	hipMemcpy(d_weights, weights, sizeof(double) * EQUATION_ELEMENTS,
			hipMemcpyHostToDevice);

	//Calc blocks and threads
	int block_size = 256;
	int grid_size = (possibleOffset + block_size - 1) / block_size;

	comparisonMutantPrime_gpu<<<grid_size,block_size>>>(d_primeSeq, d_mutant, lenMutant, possibleOffset,d_weights, d_offsetScore, my_rank);

	//Transfer data from GPU to CPU
	hipMemcpy(offsetScore, d_offsetScore, sizeof(char) * possibleOffset,
			hipMemcpyDeviceToHost);

	double bestScore = offsetScore[0];
	//p_offset = 0;
	for(int i=1;i<possibleOffset;i++){
		
		if(bestScore < offsetScore[i]){
			bestScore = offsetScore[i];
			//p_offset = i;	
		}
	}
	
	
	hipFree(d_primeSeq);
	hipFree(d_mutant);
	free(offsetScore);
	hipFree(d_offsetScore);
	return bestScore;
}
